#include "hip/hip_runtime.h"
static_assert(__cplusplus >= 201402L, "The test for using fmt in CUDA kernel requires compiling the device-side code as C++14 or later");

#include <fmt/core.h>
//#include <fmt/chrono.h>

#include <stdio.h>

__device__ void initialize(char* buffer, size_t buffer_size)
{
  memset(buffer, buffer_size - 1, '@');
  buffer[buffer_size - 1] = '\0';
}

__global__ void printing_kernel()
{
#define QUOTE(str) #str
#define EXPAND_AND_QUOTE(str) QUOTE(str)
#define BUFFER_SIZE 32
#define BUFFER_SIZE_STR EXPAND_AND_QUOTE(BUFFER_SIZE)
  constexpr const auto buffer_size {BUFFER_SIZE};
  printf("Block %u thread %u: Running fmt tests\n", blockIdx.x, threadIdx.x);
  const char* s = "The quick brown fox jumped over the lazy dog";
  char buf[buffer_size + 1];
  initialize(buf, buffer_size);
  // Can't use this function, since it outputs an std::string, and we don't have
  // std::strings in kernels. We _could_ implement a string-like class (and maybe
  // even use it inside fmt), which could be returned in kernel-side.
  // fmt::format("{}", s);
  fmt::format_to(buf, "{}", s);
//  buf[buffer_size] = '\0'; // to be on the safe side
  printf("fmt::format_to(buf, " BUFFER_SIZE_STR ", \"{}\", \"%s\") results in: %." BUFFER_SIZE_STR "s\n", s, buf);

  initialize(buf, buffer_size);
  fmt::format_to(buf, FMT_STRING("{}"), s);
//  buf[buffer_size] = '\0'; // to be on the safe side
  printf("fmt::format_to(buf, " BUFFER_SIZE_STR ", \"{}\", \"%s\") results in: %." BUFFER_SIZE_STR "s\n", s, buf);

  initialize(buf, buffer_size);
  fmt::format_to_n(buf, buffer_size, "{}", s);
//  buf[buffer_size] = '\0'; // to be on the safe side
  printf("fmt::format_to_n(buf, " BUFFER_SIZE_STR ", \"{}\", \"%s\") results in: %." BUFFER_SIZE_STR "s\n", s, buf);

  initialize(buf, buffer_size);
  fmt::format_to_n(buf, buffer_size, "I'd rather be {1} than {0}.", "right", "happy");
  printf("fmt::format_to_n(buf, " BUFFER_SIZE_STR ", \"I'd rather be {1} than {0}.\", \"right\", \"happy\") results in: %." BUFFER_SIZE_STR "s\n", buf);

  initialize(buf, buffer_size);
  auto bsv = fmt::basic_string_view<char> { "This is my string_view" };
  fmt::format_to_n(buf, buffer_size, "{}", bsv);
  printf("fmt::format_to_n(buf, " BUFFER_SIZE_STR ", \"{}\", \"fmt::basic_string_view<char> { \"This is my string_view\" }) results in: %." BUFFER_SIZE_STR "s\n", buf);

//  using namespace std::literals::chrono_literals;
//  fmt::format_to_n(buf, buffer_size, "Default format: {} {}\n", 42s, 100ms);

  // These can't be used, since they target stdout - and our converted fmt has no access
  // to stdout. We _could_ implement those parts of it so that it eventually uses
  // the kernel-accessible "printf()" command though.
  //
  //    fmt::print("{}", s);
  //    fmt::print(stdout, "{}", s);
  //    for(int i = 0; i < sizeof(buf)/ sizeof(char); i++) {
  //      printf("%02x ", (int) buf[i]);
  //    }
  //    printf("Formatting commands buf = %*s\n", (int) (sizeof(buf) / sizeof(buf[0])), buf);
}

int main(void)
{
    int threadsPerBlock = 1;
    int blocksPerGrid = 1;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    printing_kernel<<<blocksPerGrid, threadsPerBlock>>>();
    auto err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Failed launching the printing kernel: %s\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
      fprintf(stderr, "printing kernel execution failed: %s\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
}
